#include "hip/hip_runtime.h"
#include"body.h"
#include<iostream>
#include <time.h> //Used by fillRandom to get a variable seed
#include<random>
using namespace std;
#define N 1000

//adapted from: Scott Griffith's LM5 example
//date Accessed: 12/10/2018
__global__ void move(Body *a, float *t){
    int tid = blockIdx.x;
    if(tid < N){
        for(int i =0; i < N; i++){
            if(i != tid){
                a[tid].update(a[i], *t); //void update_pos(Body b, float nf, float t)
            }
        }
    }
}

//////////////////////////////////////////
// Random int Generation 
// Developed and introduced by Kent Jones 2016
//adapted from: Scott Griffith's LM1_part 4
//accessed: 12/10/2018
//////////////////////////////////////////
random_device rd;	// Used to produce a random seed
default_random_engine engine(rd()); // Use the defualt random number generator engine
std::uniform_real_distribution<> distribution(0, 1000); // Generate a uniform real distribution between 0, 1
//
//Thread-safe C++11 pseudo-random number generator
//@return		returns a random value between 0 and 1
//
double cs273_rand() {
	return distribution(engine);
}
////////////////////////////////////////////

int main(){

    Body* objects[N];
    for(int i = 0; i < N; i++){
        objects[i] = new Body(float(cs273_rand()), float(cs273_rand()));
    }

    // for(int i = 0; i < N; i++){
    //     objects[i] -> where_am_i();
    // }
    // Body* b1 = new Body(20.65, 50.55);
    // Body* b2 = new Body(150.5, 210.35);
    // Body* objects[2];
    // objects[0] = b1;
    // objects[1] = b2;


    //Body* result[N]; //Host data, a/b input data, c output data
    Body* dev_a; //Device data pointers for the main data
    float* dev_time; //device data pointer for time

    //Allocate memory on the device
    hipMalloc( (void**)&dev_a, N*sizeof(Body*));
    //hipMalloc( (void**)&dev_b, N*sizeof(Body*));
    hipMalloc( (void**)&dev_time, 1*sizeof(int));

    const float t = 0.01; //the time step in seconds
    float time_passed = 0.0;

    hipMemcpy(dev_time, time_passed, 1*sizeof(int), hipMemcpyHostToDevice);
    //add<<<1,100>>>(dev_a, dev_time);

    for( int i = 0; i < 200000; i++){
        hipMemcpy(dev_a,objects,N*sizeof(Body*),hipMemcpyHostToDevice);
        move<<<1,1024>>>(dev_a, dev_time);
        hipMemcpy(objects,dev_a,N*sizeof(Body*),hipMemcpyDeviceToHost);
        time_passed += t;
        hipMemcpy(dev_time, time_passed, 1*sizeof(int), hipMemcpyHostToDevice);
        }

    for(int i = 0; i < N; i++){
        objects[i] -> where_am_i();
        cout << endl;
        
    }
    return 0;
}