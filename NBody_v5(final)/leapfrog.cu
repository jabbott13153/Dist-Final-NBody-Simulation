#include "hip/hip_runtime.h"
#include<math.h>
#include<stdio.h>

#define G 6.67408E-11 //the gravatational constant G

//adapted from: http://www.physics.drexel.edu/~steve/Courses/Comp_Phys/Integrators/leapfrog/
//date accessed: 12/4/2018

typedef double real;

#define N 2	/* Number of second-order equations */
#define GM 1	/* Constant in Coulomb's law */


/* Determine (vector) a = dv/dt, given (vector) x */

void deriv(int n, real x[], real dvdt[])
{
    real r2 = 0.0, r3i;
    int i;

    for (i = 0; i < n; i++) r2 += x[i]*x[i];
    r3i = GM / (r2*sqrt(r2));		/* Inverse-square force law */

    for (i = 0; i < n; i++) dvdt[i] = -x[i]*r3i;
}


/* Advance the velocity by dt2 using the Euler method */

void advance_vel(int n, real x[], real v[], real dt2)
{
    real dvdt[N];
    int i;

    deriv(n, x, dvdt);

    for (i = 0; i < n; i++) v[i] += dt2*dvdt[i];
}


static real e0;

/* Calculate the energy (inverse-square case ONLY!) */

real energy(int n, real x[], real v[])
{
    return 0.5 * (v[0]*v[0] + v[1]*v[1])
			- GM / sqrt(x[0]*x[0] + x[1]*x[1]);
}


void output(int n, real x[], real v[], real t, real dt)
{
    int i;
    real vv[N], dvdt[N];

    printf("%f", t);
    for (i = 0; i < n; i++)  printf(" %f", x[i]);

    /* Synchronize the velocity if necessary */

    for (i = 0; i < n; i++)  vv[i] = v[i];
    if (t > 0) {
	deriv(n, x, dvdt);
	for (i = 0; i < n; i++)  vv[i] -= 0.5*dt*dvdt[i];
    }

    printf(" %f", energy(n, x, vv) - e0);
    printf("\n");
}

void leapfrog(int n, real x[], real v[], real* t, real dt)
{
    real dvdt[N];
    int i;

    for (i = 0; i < n; i++) x[i] += dt*v[i];
    deriv(n, x, dvdt);
    for (i = 0; i < n; i++) v[i] += dt*dvdt[i];

    *t += dt;
}


main(int argc, char** argv)
{
    /* General integrator x(t). */

    /* Default integration parameters and initial conditions: */

    real t = 0, x[N] = {1.0, 0.0}, v[N] = {0.0, 0.75};
    real t_max = 100, dt = 0.01;

    int i;

    /* Parse the argument list. */

    for (i = 0; i < argc; i++)
	if (argv[i][0] == '-') {
	    switch (argv[i][1]) {

		case 'd':	dt = atof(argv[++i]); break;
		case 't':	t_max = atof(argv[++i]); break;
		case 'v':	v[1] = atof(argv[++i]); break;
	    }
	}

    e0 = energy(N, x, v);
    output(N, x, v, t, dt);
    advance_vel(N, x, v, 0.5*dt);

    while (t < t_max) {
	leapfrog(N, x, v, &t, dt);
	output(N, x, v, t, dt);
    }
}